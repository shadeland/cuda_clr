#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
 * example application.
 * Host code.
 */

/*
 400 Samples 
 8192 vars 
 25 bins 
 8X8 blocksize , 128 batches 
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//this change
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
// helper functions for SDK examples
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "template_cpu.h"
#include "InfoKit2.h"
#include "FileUtil.h"
// #include <fstream>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
#define CNUMSAMPLES 400 //
#define CNUMVARS 128	// powers of two for now
#define CNUMMI NUMVARS *NUMVARS
#define CNUMBINS 25
#define CBATCHSIZE 128 //128*128 batch size for hist mem management
#define CTPBX 16	   //threads per block dim 16*16
#define CTOTAL NUMSAMPLES *NUMVARS *NUMBINS

int NUMSAMPLES;
int NUMVARS;
int NUMMI;
int NUMBINS = -1;
int BATCHSIZE;
int TPBX;
int TOTAL;
int SPLINEORDER;
int V;
char *FILENAME; 
FILE *fp;

////////////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////////////

__device__ float scale(int i, int n)
{
	return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1) * (x2 - x1));
}

//this just uses 1dim blocks
__global__ void histo2dGlobal(float *d_out, float *d_w, float *d_hist2d, float *d_entropies1d,
							  dim3 curBatch, int numBins, int numSamples, int BATCHSIZE, int NUMVARS)
{

	const int totalThreads = gridDim.x * blockDim.x; // whic is actually numvars*numvars;

	const int curMiX = blockIdx.x * blockDim.x + threadIdx.x;
	const int curMiY = blockIdx.y * blockDim.y + threadIdx.y;
	const int globalMiX = BATCHSIZE * curBatch.x + curMiX; //global MI
	const int globalMiY = BATCHSIZE * curBatch.y + curMiY;
	if ((globalMiY > globalMiX) || (globalMiY >= NUMVARS) || (globalMiX >= NUMVARS))
		return;
	//	printf("%d x %d y \n", globalMiX, globalMiY);
	//	printf("%d batch.x %d batch.y \n", curBatch.x, curBatch.y);
	int histSize = numBins * numBins;

	float temp = 0;
	int curVarXWeightStart = globalMiX * numSamples * numBins;
	int curVarYWeightStart = globalMiY * numSamples * numBins;

	int curHistStart = ((BATCHSIZE * curMiX) + curMiY) * (numBins * numBins);

	for (int curBinX = 0; curBinX < numBins; ++curBinX)
	{
		for (int curBinY = 0; curBinY < numBins; ++curBinY)
		{
			for (int curSample = 0; curSample < numSamples; ++curSample)
			{
				temp += (d_w[curVarXWeightStart + (curBinX * numSamples) + curSample] * d_w[curVarYWeightStart + (curBinY * numSamples) + curSample]) / numSamples;
				//				printf("%d bx, %d by, %d s, %d mx, %d my, %0.2f wx, %0.2f wy, %0.2f temp \n",curBinX,curBinY,curSample,globalMiX,globalMiY, d_w[curVarXWeightStart + (curBinX * numBins) + curSample] ,
				//				 d_w[curVarYWeightStart + (curBinY * numBins) + curSample], temp);
			}
			d_hist2d[curHistStart + (curBinX * numBins) + curBinY] = temp;
			//			printf("%0.2f h2d \n",  d_hist2d[curHistStart + curBinX * numBins + curBinY]);
			temp = 0;
		}
	}

	//Calc entropy on h2d
	float incr = 0;
	float H2D = 0;
	for (int curBinX = 0; curBinX < numBins; ++curBinX)
	{
		for (int curBinY = 0; curBinY < numBins; ++curBinY)
		{
			incr = (float)d_hist2d[curHistStart + (curBinX * numBins) + curBinY];
			//			printf("%0.2f incr \n",  d_hist2d[curHistStart + (curBinX * numBins) + curBinY]);
			if (incr > 0)
			{
				H2D -= incr * log2(incr); //calc entropy of current MI
			}
		}
	}
	float H1X = d_entropies1d[globalMiX];
	float H1Y = d_entropies1d[globalMiY];
	// printf("%0.2f, %0.2f \n",  H1X , H1Y);

	float MI = H1X + H1Y - H2D;

	// __syncthreads();
	d_out[(NUMVARS * globalMiX) + globalMiY] = MI;
	d_out[(NUMVARS * globalMiY) + globalMiX] = MI;
	//	printf("%d OUT %0.2f H2D",NUMVARS * globalMiX + globalMiY, H2D);
}

//////////// for benchmarking with CPU use template source
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

void getRandomData(float *w, int numSamples, int numVars, int numBins)
{

	randomI(w, numSamples * numVars * numBins);
}

// TO RUN EACH BATCH
void runBatch(int batchX, int batchY, float *d_w, float *d_out, float *d_entropies1d, float *d_hist2d, int numVars)
{
	int startVarX = batchX * BATCHSIZE; //for memory indexing
	int startVarY = batchY * BATCHSIZE;
	int endVarX = startVarX + BATCHSIZE;
	int endVarY = startVarY + BATCHSIZE;

	StopWatchInterface *timer = 0; // This can be shared
	sdkCreateTimer(&timer);

	dim3 curBatch(batchX, batchY);

	dim3 threadsPerBlock(TPBX, TPBX);
	dim3 blocksPerGrid((BATCHSIZE + TPBX - 1) / TPBX, (BATCHSIZE + TPBX - 1) / TPBX);
	if (V >= 2)
		printf("Start Runing batch (%d,%d)  \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize \n\n", batchX, batchY, NUMSAMPLES, numVars, NUMBINS, TPBX, TPBX);
	sdkStartTimer(&timer);
	histo2dGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_w, d_hist2d, d_entropies1d, curBatch, NUMBINS, NUMSAMPLES, BATCHSIZE, NUMVARS);
	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	if (V >= 2)
		printf("Processing time GPU for batch: %f (ms)\n", sdkGetTimerValue(&timer));
}

void _clacNumBinsint(float *data, int numVars, int numSamples, float binMultiplier)
{
	if (NUMBINS != -1)
	{
		return;
	}
	int *binCount ;
	binCount = calcNumBins(data, numVars, numSamples, binMultiplier);
	if (NUMBINS == -1)
	{
		NUMBINS = (int)floor(mediani(binCount, numVars));
		float stdBinCount = stdi(binCount, numVars);
		printf("Bin count not supplied. Autodetected that dataset warrants %d bins (median); stddev == %f\n", NUMBINS, stdBinCount);
		if (NUMBINS > 15)
			fprintf(stdout, "Warning: this automatic bin count (%d) may be a bit slow on large datasets, and may warrant a spline degree above 3\n", NUMBINS);
		if (NUMBINS < 2)
		{
			fprintf(stderr, "Too few bins (%d)!\n", NUMBINS);
			exit(-1);
		}
	}
}

void clac_numbins_entropies_wights(float *data, float *entropies, float *w)
{
	StopWatchInterface *timer = 0; // This can be shared
	sdkCreateTimer(&timer);

	float *knots = (float *)calloc(NUMBINS + SPLINEORDER, sizeof(float));
	const float *hist1 = (float *)calloc(NUMBINS, sizeof(float));
	float *e2d = (float *)calloc(NUMVARS * NUMVARS, sizeof(float));
	float *miMat = (float *)calloc(NUMVARS * NUMVARS, sizeof(float));
	float *miClrMat = (float *)calloc(NUMVARS * NUMVARS, sizeof(float));
	////CALC KNOTS
	knotVector(knots, NUMBINS, SPLINEORDER);

	const float *knotsC = knots;

	for (int i = 0; i < NUMVARS; i++)
	{
		findWeights(data + (i * NUMSAMPLES), knotsC, w + i * NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS, -1, -1);
		entropies[i] = entropy1d(data + i * NUMSAMPLES, knotsC, w + i * NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS);

		//RUn on cpy of test
		// sdkStartTimer(&timer);
		// for(int j=0; j<NUMVARS; j++){
		// 	e2d[i*NUMVARS+j] = entropy2d(data + (i*NUMSAMPLES), data + (j*NUMSAMPLES), knotsC, w + i*NUMSAMPLES * NUMBINS, w + j*NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS);
		// }
		// sdkStopTimer(&timer);
	}

	//calc mi clr on cpu
	sdkStartTimer(&timer);
	miSubMatrix(data, miMat, NUMBINS, NUMVARS, NUMSAMPLES, SPLINEORDER, 0, NUMVARS-1);
	clrUnweightedStouffer(miMat, miClrMat, NUMVARS);
	sdkStopTimer(&timer);
	miMat = transpose(miMat, NUMVARS, NUMVARS);
	printf("Processing time CPU : %f(ms)\n", sdkGetTimerValue(&timer));
	if (V >= 1)
	{
		fp = fopen("logcpu", "w+");
		fprintMat(fp, miClrMat, "MI CLR CPU", NUMVARS, NUMVARS);
		fclose(fp);
	}
}

void loadCsv(char *filename, float *data, char* sparator)
{
	
	int curRow = 0;
	int curCol = 0;
	gzFile input;

	char *fileBuf, **lines, **linesTf, *headSep, *curSep, *curLine, *curLineTf, **rowNames;
	char *dup; 
	int fLength;
	int numSamples, numVars,f =0;
	/* returns the length of the file.  This is needed to initiate the memory for the filebuffer.
     Also returns the number of samples (conditions) and the number of variables (genes)*/
  fLength = sizeArray(filename, &numSamples, &numVars);
  numSamples--; /* subtract 1 to account for the first row in the file which is the row of gene id */
  printf("There are %d genes and %d experimental conditions in file %s\n", numVars, numSamples, filename);

  /* initiate the fileBuffer and read the entire file in to the fileBuffer*/
  input = gzopen(filename, "rb");
  fileBuf = (char *)calloc(fLength + 1, sizeof(char));
  /* BUG FIXED: X and lines are uninitialized */
  if (fileBuf == NULL) /* || X == NULL || lines == NULL) */
  {
    fprintf(stderr, "Cannot allocate a buffer of size %ld bytes for file reading.\n", fLength + 1);
    return ;
  }
  gzread(input, fileBuf, fLength); /* Read the entire file into fileBuf */
  gzclose(input);

  /* the fileBuffer is a pointer, so to get at each line, you have to tokenize and create a pointer to a pointer with lines
     The + 1 is to take account for the first row. */
  lines = (char **)calloc((numSamples+1), sizeof(char *));

  /* I was trying the same idea as with lines to create a pointer to the pointer containing the gene names */
  rowNames = (char **)calloc(numVars, sizeof(char *));

  /* break the fileBuffer up by new lines so "lines" now contains each individual row in the file */
  curLine = strtok(fileBuf, "\n");
  
  while (curLine != NULL)
  {
    lines[f] = curLine;
    curLine = strtok(NULL, "\n");
    ++f;
  }

  /* parse the values and assingn them to X, rember that samples are rows and genes are columns */
  
  for (int l = 1; l < (numSamples + 1); ++l)
  {
    f = l;
    curSep = strtok(lines[l], sparator); /* for each line, break apart the line on the tabs */
    while (curSep != NULL)
    {
      --f;
      data[f] = atof(curSep); /* since one column is one gene, have to do a little math to get the indexes correct*/
      f += numSamples + 1;
      curSep = strtok(NULL, sparator);
    }
  }

  /* in the end, X should contain a flattened matrix of gene expression values, where the values for gene 2 are placed sequentially
     after gene 1, and gene 3 after gene 2, etc. */

  /* tokenize the first row to get the gene ids into "rowNames" */
  headSep = strtok(lines[0], sparator);
  f = 0;
  while (headSep != NULL)
  {
    rowNames[f] = headSep;
    ++f;
    headSep = strtok(NULL, sparator);
  }

  free(lines);


	// while (std::getline(in, line))
	// {	
	// 	dup = strdup(line.c_str());
	// 	curSep = strtok(dup, sparator); /* for each line, break apart the line on the comma */
	// 	while (curSep != NULL)
	// 	{
	// 		data[curRow*NUMVARS+curCol] = atof(curSep); /* since one column is one gene, have to do a little math to get the indexes correct*/
	// 		curSep = strtok(NULL, sparator);
	// 		curCol++;
	// 	}
	// 	curRow++;
	// }
}

int main(int argc, char **argv)
{
	if (argc < 2)
	{
		printf("usage: template <numSamples> <numVars> <numBins=-1> <batchSize> <threadperblock> <VEROBOSE=0||1>");
		return 1;
	}
	NUMSAMPLES = atoi(argv[1]); //
	NUMVARS = atoi(argv[2]);	// powers of two for now
	NUMMI = NUMVARS * NUMVARS;
	NUMBINS = atoi(argv[3]);
	FILENAME = argv[7];
	BATCHSIZE = atoi(argv[4]); //128*128 batch size for hist mem management
	TPBX = atoi(argv[5]);	  //threads per block dim 16*16
	V = atoi(argv[6]);
	SPLINEORDER = 3;

	// if (V >= 1)
	// 	fp = fopen("log", "w+");

	// Declare a pointer for an array of floats
	float *h_out = 0;
	float *d_out = 0;
	float *h_w = 0;
	float *d_w = 0;
	float *h_entrop1d = 0;
	float *d_entropies1d = 0;
	float *h_data = 0;
	float *d_hist2d = 0;
	float *h_clrMat = 0;

	// setup a time to calc the time
	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);

	h_data = (float *)calloc(NUMVARS * NUMSAMPLES, sizeof(float));
	h_out = (float *)calloc(NUMMI, sizeof(float));
	h_entrop1d = (float *)calloc(NUMVARS, sizeof(float));
	h_clrMat = (float *)calloc(NUMVARS * NUMVARS, sizeof(float));

	printf("Reading %s File:",FILENAME);
	

	// getRandomData(h_data, NUMSAMPLES, NUMVARS, 1); // generate random data

	
	//Read Data from csv file 
	loadCsv(FILENAME, h_data, ",");
	// printMat(h_data, "DATA", NUMVARS, NUMSAMPLES);
	// calc num bins
	_clacNumBinsint(h_data, NUMVARS, NUMSAMPLES, 1);

	TOTAL = NUMSAMPLES * NUMVARS * NUMBINS;
	h_w = (float *)calloc(TOTAL, sizeof(float)); // host mem for weights /// why float ? ???

	// Allocate device memory to store the output array with size number  samples
	// 1d for now
	hipMalloc(&d_out, NUMMI * sizeof(float));
	hipMemset(d_out, 0, NUMMI * sizeof(float));
	hipMalloc(&d_entropies1d, NUMVARS * sizeof(float));
	hipMalloc(&d_w, TOTAL * sizeof(float));
	hipMalloc(&d_hist2d, NUMBINS * NUMBINS * BATCHSIZE * BATCHSIZE * sizeof(float));

	// gen random data

	if (V >= 3)
		fprintMat(fp, h_data, "DATA MAT", NUMVARS, NUMSAMPLES);

	clac_numbins_entropies_wights(h_data, h_entrop1d, h_w);

	if (V >= 3)
		fprintMat(fp, h_entrop1d, "ENTROPY1 MAT", NUMVARS, 1);

	if (V >= 3)
		fprintMat(fp, h_w, "WEIGHT MAT", NUMVARS, NUMSAMPLES * NUMBINS);

	//copy w to dev
	// copy result entropy to gpu
	hipMemcpy(d_w, h_w, TOTAL * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_entropies1d, h_entrop1d, NUMVARS * sizeof(float), hipMemcpyHostToDevice);

	// runing batches

	int numBatches = (NUMVARS + BATCHSIZE - 1) / BATCHSIZE;
	sdkStartTimer(&timer);
	for (int curBatchX = 0; curBatchX < numBatches; ++curBatchX)
	{

		for (int curBatchY = 0; curBatchY < numBatches; ++curBatchY)
		{
			runBatch(curBatchX, curBatchY, d_w, d_out, d_entropies1d, d_hist2d, BATCHSIZE);
		}
	}
	sdkStopTimer(&timer);
	hipMemcpy(h_out, d_out, NUMMI * sizeof(float), hipMemcpyDeviceToHost);

	if (V >= 1)
	{
		fp = fopen("loggpu", "w+");
		fprintMat(fp, h_out, "ENTROPY2 MAT", NUMVARS, NUMVARS);
		fclose(fp);
	}
	// Launch kernel to compute and store distance values
	// printf("Start Runing \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize \n\n",NUMSAMPLES , NUMVARS , NUMBINS , TPBX,TPBX);
	// sdkStartTimer(&timer);
	// histo2dGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_w, d_hist2d, NUMBINS, NUMSAMPLES);
	// hipDeviceSynchronize();
	// sdkStopTimer(&timer);
	printf("Finished Runing  \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize , %d batches \n\n",
		   NUMSAMPLES, NUMVARS, NUMBINS, TPBX, TPBX, BATCHSIZE);
	printf("Processing Total Time GPU: %f (ms)\n", sdkGetTimerValue(&timer));

	///Done with the gpu??? ?

	/// generate CLRWeighted
	// it needs to be transposed??
	// h_out = transpose(h_out, NUMVARS, NUMVARS);
	clrUnweightedStouffer(h_out, h_clrMat, NUMVARS);

	if (V >= 1)
	{
		fp = fopen("loggpu", "w+");
		fprintMat(fp, h_clrMat, "CLR MAT", NUMVARS, NUMVARS);
		fclose(fp);
	}
	// hipMemcpy(h_out, d_out, NUMMI*sizeof(float), hipMemcpyDeviceToHost);

	/************************************************************************************************
   get and sort the output */

	/***** A more efficient way to do it would be as in the Java example "BasicCorrelation" *****/

	/* from the CLR matrix, select out all TF to target gene interactions and place them into S */
	// float *C = h_clrMat;
	// int numPossibleEdges = (NUMVARS * NUMVARS);
	// float *S = (float *)calloc(numPossibleEdges, sizeof(float));
	// int i, j, l, f;
	// f = 0;
	// for (i = 0; i < NUMVARS; i++)
	// {
	// 	for (j = 0; j < NUMVARS; j++)
	// 	{
	// 		if (i != j)
	// 		{
	// 			S[f] = C[i * NUMVARS + j];
	// 			++f;
	// 		}
	// 	}
	// }

	// qsort(S, numPossibleEdges, sizeof(float), compare_floats);

	// int CUT = numPossibleEdges;

	// f = 0;

	// /* loop through the ranked set of CLR values and print out any  TF to target prediction
    //  that matches the current CLR score.  Stop at 100,000 and print to the output file. */

	// FILE *out = fopen("logedges", "w");
	// float prev = 0;
	// for (int l = numPossibleEdges - 1; l > 0; --l)
	// {
	// 	if (S[l] != prev)
	// 	{ /* make sure we are not repeating predictions. this will avoid printing predictions more than once */
	// 		prev = S[l];
	// 		for (i = 0; i < NUMSAMPLES; i++)
	// 		{
	// 			for (j = 0; j < NUMVARS; j++)
	// 			{
	// 				if (i != j)
	// 				{
	// 					if (C[i * NUMVARS + j] == S[l])
	// 					{
	// 						fprintf(out, "%d\t%d\t%f\n", i, j, C[i * NUMVARS + j]);
	// 						++f;
	// 						/* If the cutoff is reached, break the three loops */
	// 						if (f >= CUT)
	// 						{
	// 							j = NUMVARS;
	// 							i = NUMVARS;
	// 							l = -1;
	// 						}
	// 					}
	// 				}
	// 			}
	// 		}
	// 	}
	// }

	hipFree(d_out); // Free the memory
	return 0;
}
////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
