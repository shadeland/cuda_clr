#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
 * example application.
 * Host code.
 */

/*
 400 Samples 
 8192 vars 
 25 bins 
 8X8 blocksize , 128 batches 
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//this change
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
// helper functions for SDK examples
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "template_cpu.h"
#include "InfoKit2.h"
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
#define CNUMSAMPLES 400 //
#define CNUMVARS 128	// powers of two for now
#define CNUMMI NUMVARS *NUMVARS
#define CNUMBINS 25
#define CBATCHSIZE 128 //128*128 batch size for hist mem management
#define CTPBX 16	   //threads per block dim 16*16
#define CTOTAL NUMSAMPLES *NUMVARS *NUMBINS

int NUMSAMPLES;
int NUMVARS;
int NUMMI;
int NUMBINS;
int BATCHSIZE;
int TPBX;
int TOTAL;
int SPLINEORDER;
int V;
FILE *fp;

////////////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////////////

__device__ float scale(int i, int n)
{
	return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1) * (x2 - x1));
}

//this just uses 1dim blocks
__global__ void histo2dGlobal(float *d_out, float *d_w, float *d_hist2d,
							  dim3 curBatch, int numBins, int numSamples, int BATCHSIZE, int NUMVARS)
{
	/* 
		each block fits 16 hist2d so blocksize should be 4/4 but it could be calcualted
		based on the number of bins with this formula; 
		b = 48000/num_of_bins*num_of_bins*4
		
		a = a - a%b 

	*/

	//shared mem for hist2d 
	extern	__shared__ float s_hist2d[]; 

	const int totalThreads = gridDim.x * blockDim.x; // whic is actually numvars*numvars;

	const int curMiX = blockIdx.x * blockDim.x + threadIdx.x;
	const int curMiY = blockIdx.y * blockDim.y + threadIdx.y;

	const int globalMiX = BATCHSIZE * curBatch.x + curMiX; //global MI
	const int globalMiY = BATCHSIZE * curBatch.y + curMiY;

	//clac cur history array
	float *s_curHist2D = (float*) &s_hist2d[((blockDim.x*threadIdx.x)+threadIdx.y)*numBins*numBins];

	if((globalMiY>globalMiX)||(globalMiY>=NUMVARS)|| (globalMiX>=NUMVARS))
			return ;		
//	printf("%d x %d y \n", globalMiX, globalMiY);
//	printf("%d batch.x %d batch.y \n", curBatch.x, curBatch.y);
	int histSize = numBins * numBins;

	float temp = 0;
	int curVarXWeightStart = globalMiX * numSamples * numBins;
	int curVarYWeightStart = globalMiY * numSamples * numBins;

	int curHistStart = ((BATCHSIZE * curMiX) + curMiY) * (numBins * numBins);

	for (int curBinX = 0; curBinX < numBins; ++curBinX)
	{
		for (int curBinY = 0; curBinY < numBins; ++curBinY)
		{
			for (int curSample = 0; curSample < numSamples; ++curSample)
			{
				temp += (d_w[curVarXWeightStart + (curBinX * numSamples) + curSample] * d_w[curVarYWeightStart + (curBinY * numSamples) + curSample]) / numSamples;
//				printf("%d bx, %d by, %d s, %d mx, %d my, %0.2f wx, %0.2f wy, %0.2f temp \n",curBinX,curBinY,curSample,globalMiX,globalMiY, d_w[curVarXWeightStart + (curBinX * numBins) + curSample] ,
//				 d_w[curVarYWeightStart + (curBinY * numBins) + curSample], temp);
			}
			 s_curHist2D[(curBinX * numBins) + curBinY] = temp;
			// printf("%0.2f h2d \n",  s_curHist2D[(curBinX * numBins) + curBinY]);
			temp = 0;
		}
	}

	//Calc entropy on h2d
	float incr = 0;
	float H2D = 0;
	for (int curBinX = 0; curBinX < numBins; ++curBinX)
	{
		for (int curBinY = 0; curBinY < numBins; ++curBinY)
		{
			incr = (float) s_curHist2D[(curBinX * numBins) + curBinY];
//			printf("%0.2f incr \n",  d_hist2d[curHistStart + (curBinX * numBins) + curBinY]);
			if (incr > 0)
			{
				H2D -= incr * log2(incr); //calc entropy of current MI
			}
		}
	}

	// __syncthreads();
	d_out[(NUMVARS * globalMiX) + globalMiY] =  H2D;
//	printf("%d OUT %0.2f H2D",NUMVARS * globalMiX + globalMiY, H2D);
}

//////////// for benchmarking with CPU use template source
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

void genWeights(float *w, int numSamples, int numVars, int numBins)
{

	randomI(w, numSamples * numVars * numBins);
}

// TO RUN EACH BATCH
void runBatch(int batchX, int batchY, float *d_w, float *d_out, float *d_hist2d, int numVars)
{
	int startVarX = batchX * BATCHSIZE; //for memory indexing
	int startVarY = batchY * BATCHSIZE;
	int endVarX = startVarX + BATCHSIZE;
	int endVarY = startVarY + BATCHSIZE;
	int sharedSize = TPBX*TPBX*NUMBINS*NUMBINS*sizeof(float);
	StopWatchInterface *timer = 0; // This can be shared
	sdkCreateTimer(&timer);

	dim3 curBatch(batchX, batchY);

	dim3 threadsPerBlock(TPBX, TPBX);
	dim3 blocksPerGrid((BATCHSIZE + TPBX - 1) / TPBX, (BATCHSIZE + TPBX - 1) / TPBX);
	if(V>=2)
		printf("Start Runing batch (%d,%d)  \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize \n\n", batchX, batchY, NUMSAMPLES, numVars, NUMBINS, TPBX, TPBX);
	sdkStartTimer(&timer);
	histo2dGlobal<<<blocksPerGrid, threadsPerBlock,sharedSize>>>(d_out, d_w, d_hist2d, curBatch, NUMBINS, NUMSAMPLES, BATCHSIZE, NUMVARS);
	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	if(V>=2)
		printf("Processing time GPU for batch: %f (ms)\n", sdkGetTimerValue(&timer));
}


void clac_numbins_entropies_wights(float *data, float *entropies, float *w)
{
	StopWatchInterface *timer = 0; // This can be shared
	sdkCreateTimer(&timer);

	float *knots = (float *)calloc(NUMBINS + SPLINEORDER, sizeof(float));
	const float *hist1 = (float *)calloc(NUMBINS, sizeof(float));
	float *e2d = (float *)calloc(NUMVARS *NUMVARS, sizeof(float));
	////CALC KNOTS
	knotVector(knots, NUMBINS, SPLINEORDER);

	const float *knotsC = knots;

	for (int i = 0; i < NUMVARS; i++)
	{
		findWeights(data + (i * NUMSAMPLES), knotsC, w + i * NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS, -1, -1);
		entropies[i] = entropy1d(data +i*NUMSAMPLES, knotsC, w + i * NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS);

		// RUn on cpu for test
		// TODO use flag
		sdkStartTimer(&timer);
		for(int j=0; j<NUMVARS; j++){
			e2d[i*NUMVARS+j] = entropy2d(data + (i*NUMSAMPLES), data + (j*NUMSAMPLES), knotsC, w + i*NUMSAMPLES * NUMBINS, w + j*NUMSAMPLES * NUMBINS, NUMSAMPLES, SPLINEORDER, NUMBINS);
		}
		sdkStopTimer(&timer);
	}
	
	printf("Processing time CPU : %f(ms)\n", sdkGetTimerValue(&timer));
	if(V >= 1){
		fp = fopen("logcpu","w+");
		fprintMat(fp,e2d, "ENTROPY 2D", NUMVARS, NUMVARS);
		fclose(fp);
	}
}




int main(int argc, char **argv)
{
	if (argc < 2)
	{
		printf("usage: template <numSamples> <numVars> <numBins> <batchSize> <threadperblock> <VEROBOSE=0||1>");
		return 1;
	}
	NUMSAMPLES = atoi(argv[1]); //
	NUMVARS = atoi(argv[2]);	// powers of two for now
	NUMMI = NUMVARS * NUMVARS;
	NUMBINS = atoi(argv[3]);
	BATCHSIZE = atoi(argv[4]); //128*128 batch size for hist mem management
	TPBX = atoi(argv[5]);	  //threads per block dim 16*16
	V = atoi(argv[6]);
	SPLINEORDER = 3;
	TOTAL = NUMSAMPLES * NUMVARS * NUMBINS;
	

	// if (V >= 1)
	// 	fp = fopen("log", "w+");

	// Declare a pointer for an array of floats
	float *h_out = 0;
	float *d_out = 0;
	float *h_w = 0;
	float *d_w = 0;
	float *h_entrop1d = 0;
	float *d_entrop1d = 0;
	float *h_data = 0;
	float *d_hist2d = 0;

	// setup a time to calc the time
	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);

	// Allocate device memory to store the output array with size number  samples
	// 1d for now
	hipMalloc(&d_out, NUMMI * sizeof(float));
	hipMalloc(&d_w, TOTAL * sizeof(float));
	hipMalloc(&d_hist2d, NUMBINS * NUMBINS * BATCHSIZE * BATCHSIZE * sizeof(float));

	h_out = (float *)calloc(NUMMI, sizeof(float));
	h_w = (float *)calloc(TOTAL, sizeof(float)); // host mem for weights /// why float ? ???
	h_data = (float *)calloc(NUMVARS * NUMSAMPLES, sizeof(float));
	h_entrop1d = (float *)calloc(NUMVARS, sizeof(float));

	// gen random data
	genWeights(h_data, NUMSAMPLES, NUMVARS, 1);

	if (V >= 3)
		fprintMat(fp, h_data, "DATA MAT", NUMVARS, NUMSAMPLES);

	clac_numbins_entropies_wights(h_data, h_entrop1d, h_w);

	if (V >= 3)
		fprintMat(fp, h_entrop1d, "ENTROPY1 MAT", NUMVARS, 1);

	if (V >= 3)
		fprintMat(fp, h_w, "WEIGHT MAT", NUMVARS, NUMSAMPLES * NUMBINS);

	//copy w to dev
	hipMemcpy(d_w, h_w, TOTAL * sizeof(float), hipMemcpyHostToDevice);

	//config kernel

	// runing batches

	int numBatches = (NUMVARS + BATCHSIZE - 1) / BATCHSIZE;
	sdkStartTimer(&timer);
	for (int curBatchX = 0; curBatchX < numBatches; ++curBatchX)
	{

		for (int curBatchY = 0; curBatchY < numBatches; ++curBatchY)
		{
			runBatch(curBatchX, curBatchY, d_w, d_out, d_hist2d, BATCHSIZE);
		}
	}
	sdkStopTimer(&timer);
	hipMemcpy(h_out, d_out, NUMMI * sizeof(float), hipMemcpyDeviceToHost);

	if (V >= 1){
		fp = fopen("loggpu","w+");
		fprintMat(fp, h_out, "ENTROPY2 MAT", NUMVARS, NUMVARS);
		fclose(fp);
	}
	// Launch kernel to compute and store distance values
	// printf("Start Runing \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize \n\n",NUMSAMPLES , NUMVARS , NUMBINS , TPBX,TPBX);
	// sdkStartTimer(&timer);
	// histo2dGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_w, d_hist2d, NUMBINS, NUMSAMPLES);
	// hipDeviceSynchronize();
	// sdkStopTimer(&timer);
	printf("Finished Runing  \n %d Samples \n %d vars \n %d bins \n %dX%d blocksize , %d batches \n\n",
		   NUMSAMPLES, NUMVARS, NUMBINS, TPBX, TPBX, BATCHSIZE);
	printf("Processing Total Time GPU: %f (ms)\n", sdkGetTimerValue(&timer));


	// hipMemcpy(h_out, d_out, NUMMI*sizeof(float), hipMemcpyDeviceToHost);

	//	sdkResetTimer(&timer);
	//	sdkStartTimer(&timer);
	//	distanceCpu();
	//	sdkStopTimer(&timer);
	//	printf("Processing time CPU: %f (ms)\n", sdkGetTimerValue(&timer));

	hipFree(d_out); // Free the memory
	return 0;
}
////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
